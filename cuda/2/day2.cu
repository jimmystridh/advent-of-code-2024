#include <iostream>
#include <vector>
#include <string>
#include <sstream>
#include <algorithm>
#include <hip/hip_runtime.h>

__device__ bool isSafeReport(const int* levels, int length) {
    if (length < 2) return false;

    bool allIncreasing = true;
    bool allDecreasing = true;

    for (int i = 0; i < length - 1; ++i) {
        int diff = levels[i + 1] - levels[i];
        if (diff <= 0) allIncreasing = false;
        if (diff >= 0) allDecreasing = false;
        if (abs(diff) < 1 || abs(diff) > 3) return false;
    }

    return allIncreasing || allDecreasing;
}

__global__ void analyzeReports(const int* d_levels, const int* d_indices, const int* d_lengths, int numReports, int* d_safePart1, int* d_safePart2) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numReports) return;

    const int* levels = d_levels + d_indices[idx];
    int length = d_lengths[idx];

    bool safePart1 = isSafeReport(levels, length);
    bool safePart2 = safePart1;

    if (!safePart1) {
        // Try removing one level at a time
        for (int i = 0; i < length; ++i) {
            // Create a new levels array without the i-th element
            int tempLength = length - 1;
            int tempLevels[100]; // Assuming max report length is less than 100
            int k = 0;
            for (int j = 0; j < length; ++j) {
                if (j != i) {
                    tempLevels[k++] = levels[j];
                }
            }
            if (isSafeReport(tempLevels, tempLength)) {
                safePart2 = true;
                break;
            }
        }
    }

    if (safePart1) atomicAdd(d_safePart1, 1);
    if (safePart2) atomicAdd(d_safePart2, 1);
}

int main() {
    std::vector<std::vector<int>> reports;
    std::string line;

    // Read input reports
    while (std::getline(std::cin, line)) {
        if (line.empty()) continue;

        std::istringstream iss(line);
        std::vector<int> levels;
        int num;
        while (iss >> num) {
            levels.push_back(num);
        }
        reports.push_back(levels);
    }

    int totalLevels = 0;
    for (const auto& report : reports) {
        totalLevels += report.size();
    }

    // Flatten levels and prepare indices
    std::vector<int> h_levels;
    std::vector<int> h_indices;
    std::vector<int> h_lengths;

    int index = 0;
    for (const auto& report : reports) {
        h_indices.push_back(index);
        h_lengths.push_back(report.size());
        h_levels.insert(h_levels.end(), report.begin(), report.end());
        index += report.size();
    }

    int numReports = reports.size();

    // Allocate device memory
    int* d_levels;
    int* d_indices;
    int* d_lengths;
    int* d_safePart1;
    int* d_safePart2;

    hipMalloc(&d_levels, h_levels.size() * sizeof(int));
    hipMalloc(&d_indices, h_indices.size() * sizeof(int));
    hipMalloc(&d_lengths, h_lengths.size() * sizeof(int));
    hipMalloc(&d_safePart1, sizeof(int));
    hipMalloc(&d_safePart2, sizeof(int));

    // Copy data to device
    hipMemcpy(d_levels, h_levels.data(), h_levels.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_indices, h_indices.data(), h_indices.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_lengths, h_lengths.data(), h_lengths.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_safePart1, 0, sizeof(int));
    hipMemset(d_safePart2, 0, sizeof(int));

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numReports + threadsPerBlock - 1) / threadsPerBlock;

    analyzeReports<<<blocksPerGrid, threadsPerBlock>>>(d_levels, d_indices, d_lengths, numReports, d_safePart1, d_safePart2);
    hipDeviceSynchronize();

    // Copy results back to host
    int h_safePart1 = 0;
    int h_safePart2 = 0;

    hipMemcpy(&h_safePart1, d_safePart1, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_safePart2, d_safePart2, sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_levels);
    hipFree(d_indices);
    hipFree(d_lengths);
    hipFree(d_safePart1);
    hipFree(d_safePart2);

    // Output results
    std::cout << "Part One: " << h_safePart1 << std::endl;
    std::cout << "Part Two: " << h_safePart2 << std::endl;

    return 0;
}
