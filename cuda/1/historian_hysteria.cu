#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <unordered_map>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/unique.h>
#include <thrust/gather.h>
#include <thrust/binary_search.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>

struct abs_diff
{
    __host__ __device__
    int operator()(const int& a, const int& b) const
    {
        return abs(a - b);
    }
};

// Functor to replace the lambda
struct MapCountsFunctor
{
    int* d_indices;
    int* d_unique_keys;
    int* d_counts;
    int num_unique;
    int* d_left;

    MapCountsFunctor(int* indices, int* unique_keys, int* counts, int num_unique, int* left)
        : d_indices(indices), d_unique_keys(unique_keys), d_counts(counts), num_unique(num_unique), d_left(left) {}

    __device__
    int operator()(int i) const
    {
        int idx = d_indices[i];
        if (idx < num_unique && d_unique_keys[idx] == d_left[i])
        {
            return d_counts[idx];
        }
        else
        {
            return 0;
        }
    }
};

int main()
{
    // Read input data
    std::vector<int> left_list;
    std::vector<int> right_list;

    int left_num, right_num;
    while (std::cin >> left_num >> right_num)
    {
        left_list.push_back(left_num);
        right_list.push_back(right_num);
    }

    int N = left_list.size();

    // Transfer data to device (GPU)
    thrust::device_vector<int> d_left(left_list);
    thrust::device_vector<int> d_right(right_list);

    // Part One: Total Distance Calculation
    // Sort both lists on the device
    thrust::sort(d_left.begin(), d_left.end());
    thrust::sort(d_right.begin(), d_right.end());

    // Compute absolute differences in parallel
    thrust::device_vector<int> d_differences(N);
    thrust::transform(d_left.begin(), d_left.end(), d_right.begin(), d_differences.begin(), abs_diff());

    // Compute the total distance using parallel reduction
    int total_distance = thrust::reduce(d_differences.begin(), d_differences.end(), 0, thrust::plus<int>());

    std::cout << "Total distance: " << total_distance << std::endl;

    // Part Two: Similarity Score Calculation

    // Step 1: Sort the right list to prepare for counting
    thrust::sort(d_right.begin(), d_right.end());

    // Step 2: Count the occurrences of each unique number in the right list
    thrust::device_vector<int> d_unique_keys(N);
    thrust::device_vector<int> d_counts(N);

    auto new_end = thrust::reduce_by_key(
        d_right.begin(), d_right.end(),
        thrust::constant_iterator<int>(1),
        d_unique_keys.begin(),
        d_counts.begin()
    );

    int num_unique = new_end.first - d_unique_keys.begin();

    // Resize vectors to the number of unique elements
    d_unique_keys.resize(num_unique);
    d_counts.resize(num_unique);

    // Step 3: Map counts to the left list numbers
    // For each element in d_left, find its index in d_unique_keys
    // Use thrust::lower_bound since d_unique_keys is sorted
    thrust::device_vector<int> d_indices(N);
    thrust::lower_bound(
        d_unique_keys.begin(), d_unique_keys.end(),
        d_left.begin(), d_left.end(),
        d_indices.begin()
    );

    // Create a functor to map counts
    MapCountsFunctor map_counts(
        thrust::raw_pointer_cast(d_indices.data()),
        thrust::raw_pointer_cast(d_unique_keys.data()),
        thrust::raw_pointer_cast(d_counts.data()),
        num_unique,
        thrust::raw_pointer_cast(d_left.data())
    );

    // Create a device vector to hold counts corresponding to the left list numbers
    thrust::device_vector<int> d_left_counts(N);

    // Use thrust::transform with the functor
    thrust::transform(
        thrust::make_counting_iterator(0),
        thrust::make_counting_iterator(N),
        d_left_counts.begin(),
        map_counts
    );

    // Compute the similarity score
    // For each number in the left list, multiply it by its count in the right list
    thrust::device_vector<int> d_similarity(N);
    thrust::transform(
        d_left.begin(), d_left.end(),
        d_left_counts.begin(),
        d_similarity.begin(),
        thrust::multiplies<int>()
    );

    // Sum up the similarity scores
    int similarity_score = thrust::reduce(d_similarity.begin(), d_similarity.end(), 0, thrust::plus<int>());

    std::cout << "Similarity score: " << similarity_score << std::endl;

    return 0;
}