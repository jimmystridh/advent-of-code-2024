// Include necessary headers

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// Define grid dimensions (maximum expected sizes)
#define MAX_ROWS 1024
#define MAX_COLS 1024

// Define the target word for Part One
#define WORD "XMAS"
#define WORD_LENGTH 4

// Device function to check if a position is within grid bounds
__device__ bool isValid(int x, int y, int numRows, int numCols) {
    return (x >= 0 && x < numRows && y >= 0 && y < numCols);
}

// Kernel for Part One
__global__ void countXMAS(char *grid, int numRows, int numCols, int *count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int totalCells = numRows * numCols;

    if (idx >= totalCells) return;

    int x = idx / numCols;
    int y = idx % numCols;

    // Define directions: N, NE, E, SE, S, SW, W, NW
    int directions[8][2] = { {-1, 0}, {-1, 1}, {0, 1}, {1, 1},
                             {1, 0}, {1, -1}, {0, -1}, {-1, -1} };

    for (int dir = 0; dir < 8; dir++) {
        bool match = true;
        int dx = directions[dir][0];
        int dy = directions[dir][1];
        int xi = x, yi = y;

        for (int i = 0; i < WORD_LENGTH; i++) {
            if (!isValid(xi, yi, numRows, numCols) || grid[xi * numCols + yi] != WORD[i]) {
                match = false;
                break;
            }
            xi += dx;
            yi += dy;
        }
        if (match) {
            atomicAdd(count, 1);
        }
    }
}

// Kernel for Part Two
__global__ void countXMAS_X(char *grid, int numRows, int numCols, int *count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int totalCells = (numRows - 2) * (numCols - 2); // Adjust for 3x3 pattern

    if (idx >= totalCells) return;

    int x = idx / (numCols - 2);
    int y = idx % (numCols - 2);

    // Positions in the X shape
    int positions[5][2] = {
        {x, y},         // Top-left or top-right
        {x + 1, y + 1}, // Center
        {x + 2, y + 2}, // Bottom-right or bottom-left
        {x + 2, y},     // Bottom-left or bottom-right
        {x, y + 2}      // Top-right or top-left
    };

    // Extract letters at the positions
    char diag1[3]; // Diagonal from top-left to bottom-right
    char diag2[3]; // Diagonal from top-right to bottom-left

    diag1[0] = grid[positions[0][0] * numCols + positions[0][1]];
    diag1[1] = grid[positions[1][0] * numCols + positions[1][1]];
    diag1[2] = grid[positions[2][0] * numCols + positions[2][1]];

    diag2[0] = grid[positions[4][0] * numCols + positions[4][1]];
    diag2[1] = grid[positions[1][0] * numCols + positions[1][1]];
    diag2[2] = grid[positions[3][0] * numCols + positions[3][1]];

    // Valid patterns (MAS or SAM)
    const char *patterns[] = { "MAS", "SAM" };

    bool matchDiag1 = false;
    bool matchDiag2 = false;

    // Check diag1
    for (int i = 0; i < 2; i++) {
        if ((diag1[0] == patterns[i][0] && diag1[1] == patterns[i][1] && diag1[2] == patterns[i][2]) ||
            (diag1[0] == patterns[i][2] && diag1[1] == patterns[i][1] && diag1[2] == patterns[i][0])) {
            matchDiag1 = true;
            break;
        }
    }

    // Check diag2
    for (int i = 0; i < 2; i++) {
        if ((diag2[0] == patterns[i][0] && diag2[1] == patterns[i][1] && diag2[2] == patterns[i][2]) ||
            (diag2[0] == patterns[i][2] && diag2[1] == patterns[i][1] && diag2[2] == patterns[i][0])) {
            matchDiag2 = true;
            break;
        }
    }

    if (matchDiag1 && matchDiag2 && diag1[1] == 'A' && diag2[1] == 'A') {
        atomicAdd(count, 1);
    }
}

// Host code
int main() {
    // Read the grid from input
    char hostGrid[MAX_ROWS * MAX_COLS];
    int numRows = 0, numCols = 0;

    char line[MAX_COLS + 2]; // +2 for newline and null terminator
    while (fgets(line, sizeof(line), stdin)) {
        size_t len = strlen(line);
        // Remove newline character
        if (line[len - 1] == '\n') {
            line[len - 1] = '\0';
            len--;
        }
        if (numCols == 0) {
            numCols = len;
        } else if (len != numCols) {
            fprintf(stderr, "Error: Inconsistent row lengths.\n");
            return 1;
        }
        memcpy(&hostGrid[numRows * numCols], line, numCols);
        numRows++;
    }

    // Allocate device memory
    char *deviceGrid;
    int *deviceCountPart1;
    int *deviceCountPart2;
    hipMalloc((void **)&deviceGrid, numRows * numCols * sizeof(char));
    hipMalloc((void **)&deviceCountPart1, sizeof(int));
    hipMalloc((void **)&deviceCountPart2, sizeof(int));

    // Copy grid to device
    hipMemcpy(deviceGrid, hostGrid, numRows * numCols * sizeof(char), hipMemcpyHostToDevice);
    hipMemset(deviceCountPart1, 0, sizeof(int));
    hipMemset(deviceCountPart2, 0, sizeof(int));

    // Define block and grid sizes
    int totalCells = numRows * numCols;
    int blockSize = 256;
    int gridSize = (totalCells + blockSize - 1) / blockSize;

    // Launch kernel for Part One
    countXMAS<<<gridSize, blockSize>>>(deviceGrid, numRows, numCols, deviceCountPart1);

    // Adjust total cells for 3x3 grid in Part Two
    int totalCellsPart2 = (numRows - 2) * (numCols - 2);
    int gridSizePart2 = (totalCellsPart2 + blockSize - 1) / blockSize;

    // Launch kernel for Part Two
    countXMAS_X<<<gridSizePart2, blockSize>>>(deviceGrid, numRows, numCols, deviceCountPart2);

    // Copy results back to host
    int countPart1 = 0, countPart2 = 0;
    hipMemcpy(&countPart1, deviceCountPart1, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&countPart2, deviceCountPart2, sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(deviceGrid);
    hipFree(deviceCountPart1);
    hipFree(deviceCountPart2);

    // Output the results
    printf("Part One: %d\n", countPart1);
    printf("Part Two: %d\n", countPart2);

    return 0;
}