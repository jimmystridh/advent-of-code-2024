// mull_it_over.cu

#include <iostream>
#include <string>
#include <regex>
#include <vector>
#include <numeric>
#include <hip/hip_runtime.h>

// Kernel to compute products of X and Y arrays
__global__ void compute_products(int* x, int* y, int* products, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        products[idx] = x[idx] * y[idx];
    }
}

int main() {
    // Read input from standard input
    std::string input((std::istreambuf_iterator<char>(std::cin)), std::istreambuf_iterator<char>());

    // Define regex pattern to match 'mul(X,Y)', 'do()', and 'don't()' instructions
    std::regex pattern(R"(mul\(([0-9]{1,3}),([0-9]{1,3})\)|do\(\)|don't\(\))");

    // Find all matches
    std::vector<std::smatch> matches;
    auto words_begin = std::sregex_iterator(input.begin(), input.end(), pattern);
    auto words_end = std::sregex_iterator();

    for (std::sregex_iterator i = words_begin; i != words_end; ++i) {
        matches.push_back(*i);
    }

    // Vectors to store X and Y values for Part One
    std::vector<int> x_values;
    std::vector<int> y_values;

    // Part Two variables
    bool mulEnabled = true;
    std::vector<int> x_enabled;
    std::vector<int> y_enabled;

    // Process matches
    for (const auto& match : matches) {
        std::string value = match.str();
        if (value == "do()") {
            mulEnabled = true;
        } else if (value == "don't()") {
            mulEnabled = false;
        } else if (value.substr(0, 3) == "mul") {
            int x = std::stoi(match[1]);
            int y = std::stoi(match[2]);
            x_values.push_back(x);
            y_values.push_back(y);
            if (mulEnabled) {
                x_enabled.push_back(x);
                y_enabled.push_back(y);
            }
        }
    }

    // Part One: Compute total sum of all products
    int n = x_values.size();
    int* d_x;
    int* d_y;
    int* d_products;
    int* products = new int[n];

    hipMalloc((void**)&d_x, n * sizeof(int));
    hipMalloc((void**)&d_y, n * sizeof(int));
    hipMalloc((void**)&d_products, n * sizeof(int));

    hipMemcpy(d_x, x_values.data(), n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y_values.data(), n * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    compute_products<<<gridSize, blockSize>>>(d_x, d_y, d_products, n);

    hipMemcpy(products, d_products, n * sizeof(int), hipMemcpyDeviceToHost);

    int partOneTotal = std::accumulate(products, products + n, 0);

    // Part Two: Compute total sum of enabled products
    int m = x_enabled.size();
    int* d_x_enabled;
    int* d_y_enabled;
    int* d_products_enabled;
    int* products_enabled = new int[m];

    if (m > 0) {
        hipMalloc((void**)&d_x_enabled, m * sizeof(int));
        hipMalloc((void**)&d_y_enabled, m * sizeof(int));
        hipMalloc((void**)&d_products_enabled, m * sizeof(int));

        hipMemcpy(d_x_enabled, x_enabled.data(), m * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_y_enabled, y_enabled.data(), m * sizeof(int), hipMemcpyHostToDevice);

        int gridSizeEnabled = (m + blockSize - 1) / blockSize;
        compute_products<<<gridSizeEnabled, blockSize>>>(d_x_enabled, d_y_enabled, d_products_enabled, m);

        hipMemcpy(products_enabled, d_products_enabled, m * sizeof(int), hipMemcpyDeviceToHost);
    }

    int partTwoTotal = std::accumulate(products_enabled, products_enabled + m, 0);

    // Free device memory
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_products);
    if (m > 0) {
        hipFree(d_x_enabled);
        hipFree(d_y_enabled);
        hipFree(d_products_enabled);
    }

    // Free host memory
    delete[] products;
    delete[] products_enabled;

    // Output the results
    std::cout << partOneTotal << std::endl;
    std::cout << partTwoTotal << std::endl;

    return 0;
}